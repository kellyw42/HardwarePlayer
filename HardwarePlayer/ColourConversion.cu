
#include <hip/hip_runtime.h>
// nvcc --cubin -O3 -gencode=arch=compute_75,code=sm_75  ColourConversion.cu
// Note: path to ColourConversion.cubin is hardcoded in VideoConvert.h

extern "C" __global__ void NV12ToGrayScale(unsigned char *srcImage, size_t nSourcePitch, unsigned int *dstImage, size_t nDestPitch, int width, int height)
{
	int x, y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		//printf("%d,%d", x, y);
		return;
	}

	unsigned char lum = srcImage[y * nSourcePitch + x];

	dstImage[y * nDestPitch + x] = (0xFF << 24) | (lum << 16) | (lum << 8) | lum;
}


__device__ static void YUV2RGB(const unsigned int* yuvi, float* red, float* green, float* blue)
{
	float constHueColorSpaceMat[9] = { 1.1644f, 0.0f, 1.596f, 1.1644f, -0.3918f, -0.813f, 1.1644f, 2.0172f, 0.0f };

	float luma, chromaCb, chromaCr;

	// Prepare for hue adjustment
	luma = (float)yuvi[0];
	chromaCb = (float)((int)yuvi[1] - 512.0f);
	chromaCr = (float)((int)yuvi[2] - 512.0f);

	// Convert YUV To RGB with hue adjustment
	*red = (luma     * constHueColorSpaceMat[0]) +
		(chromaCb * constHueColorSpaceMat[1]) +
		(chromaCr * constHueColorSpaceMat[2]);

	*green = (luma     * constHueColorSpaceMat[3]) +
		(chromaCb * constHueColorSpaceMat[4]) +
		(chromaCr * constHueColorSpaceMat[5]);

	*blue = (luma     * constHueColorSpaceMat[6]) +
		(chromaCb * constHueColorSpaceMat[7]) +
		(chromaCr * constHueColorSpaceMat[8]);
}

__device__ static unsigned int RGBA_pack_10bit(float red, float green, float blue, unsigned int alpha)
{
	unsigned int ARGBpixel = 0;

	// Clamp final 10 bit results
	red = ::fmin(::fmax(red, 0.0f), 1023.f);
	green = ::fmin(::fmax(green, 0.0f), 1023.f);
	blue = ::fmin(::fmax(blue, 0.0f), 1023.f);

	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((unsigned int)blue >> 2) |
		(((unsigned int)green >> 2) << 8) |
		(((unsigned int)red >> 2) << 16) |
		(unsigned int)alpha);

	return ARGBpixel;
}

// CUDA kernel for outputting the final ARGB output from NV12

#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF

extern "C" __global__ void NV12ToARGB(const unsigned char* srcImage, size_t nSourcePitch,
	unsigned int* dstImageTop, unsigned int *dstImageBottom, size_t nDestPitch,
	unsigned int width, unsigned int height)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	const int y = blockIdx.y *  blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values

	unsigned int yuv101010Pel[2];
	yuv101010Pel[0] = (srcImage[y * nSourcePitch + x]) << 2;
	yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

	const size_t chromaOffset = nSourcePitch * height;

	int y_chroma = ((y >> 2) << 1) | (y & 1);

	unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x];
	unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

	yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
	yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
	yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

	unsigned int yuvi[6];
	float red[2], green[2], blue[2];

	yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
	yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
	yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
	yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
	yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
	yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
	YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

	unsigned int* dstImage = (y & 1) ? dstImageBottom : dstImageTop;
	int offset = (y >> 1) * width + x;
	dstImage[offset] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
	dstImage[offset + 1] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
}

extern "C" __global__ void FinishLine3(const unsigned char* srcImage, size_t nSourcePitch,
	unsigned int* dstImageTop, unsigned int *dstImageBottom, size_t nDestPitch,
	unsigned int width, unsigned int height, float angle)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int y = blockIdx.y *  blockDim.y + threadIdx.y;

	if (y >= height)
		return;

	for (int dx=0; dx<width; dx+=2)
	{
	    int x = y * angle + dx;

		unsigned int yuv101010Pel[2];
		yuv101010Pel[0] = (srcImage[y * nSourcePitch + x]) << 2;
		yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

		const size_t chromaOffset = nSourcePitch * height;

		int y_chroma = ((y >> 2) << 1) | (y & 1);

		unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x];
		unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

		yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
		yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		unsigned int yuvi[6];
		float red[2], green[2], blue[2];

		yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
		yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
		yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
		yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
		yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
		yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

		YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
		YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

		unsigned int* dstImage = (y & 1) ? dstImageBottom : dstImageTop;
		int offset = (y >> 1) * width + dx;
		dstImage[offset] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
		dstImage[offset + 1] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
	}
}

extern "C" __global__ void FinishLine2(unsigned int* dstImage, unsigned int width, unsigned int height, float top, float bottom, int y1, int y2)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || x < bottom || y < y1 || y > y2)
		return;

	float dy = ((float)y) / height;
	int copyx = dy * bottom + (1 - dy) * top;
	int src = dstImage[y * width + copyx];
	int R = ((src >> 16) & 0xFF) - 180;
	int G = ((src >> 8) & 0xFF) - 73;
	int B = ((src >> 0) & 0xFF) - 100;
	float diff = (R * R + G * G + B * B);
	if (diff < 2000)
		dstImage[y * width + x] = 0xFF000000;
	else
		dstImage[y * width + x] = 0xFFFFFFFF;
}

extern "C" __global__ void FinishLine(unsigned int* dstImage, unsigned int width, unsigned int height, float top, float bottom, int y1, int y2, int*hits)
{
	int total = 0;
	for (int y = y1; y < y2; y++)
	{
		float dy = ((float)y) / height;
		int copyx = dy * bottom + (1 - dy) * top;
		int src = dstImage[y * width + copyx];
		int R = ((src >> 16) & 0xFF) - 180;
		int G = ((src >> 8) & 0xFF) - 73;
		int B = ((src >> 0) & 0xFF) - 100;
		float diff = (R * R + G * G + B * B);
		if (diff < 2000)
			total++;
	}
	*hits = total;
}


extern "C" __global__ void Luminance(unsigned char *srcImage, size_t nSourcePitch, int left, int right, int top, int bottom, long long *lum)
{
	long long topLum = 0, bottomLum = 0;
	for (int x = left; x <= right; x++)
		for (int y = top; y <= bottom; y++)
			if (y & 1)
				bottomLum += srcImage[y * nSourcePitch + x];
			else
				topLum += srcImage[y * nSourcePitch + x];
	lum[0] = topLum;
	lum[1] = bottomLum;
}

