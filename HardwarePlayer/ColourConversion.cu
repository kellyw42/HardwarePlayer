
#include <hip/hip_runtime.h>
//nvcc --cubin -O3 -gencode=arch=compute_75,code=sm_75  ColourConversion.cu -odir ..\x64\Debug

extern "C" __global__ void NV12ToGrayScale(unsigned char *srcImage, size_t nSourcePitch, unsigned int *dstImage, size_t nDestPitch, int width, int height)
{
	int x, y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		//printf("%d,%d", x, y);
		return;
	}

	unsigned char lum = srcImage[y * nSourcePitch + x];

	dstImage[y * nDestPitch + x] = (0xFF << 24) | (lum << 16) | (lum << 8) | lum;
}

__constant__ float constHueColorSpaceMat[9] = { 1.1644f, 0.0f, 1.596f, 1.1644f, -0.3918f, -0.813f, 1.1644f, 2.0172f, 0.0f };

__device__ static void YUV2RGB(const unsigned int* yuvi, float* red, float* green, float* blue)
{
	float luma, chromaCb, chromaCr;

	// Prepare for hue adjustment
	luma = (float)yuvi[0];
	chromaCb = (float)((int)yuvi[1] - 512.0f);
	chromaCr = (float)((int)yuvi[2] - 512.0f);

	// Convert YUV To RGB with hue adjustment
	*red = (luma     * constHueColorSpaceMat[0]) +
		(chromaCb * constHueColorSpaceMat[1]) +
		(chromaCr * constHueColorSpaceMat[2]);

	*green = (luma     * constHueColorSpaceMat[3]) +
		(chromaCb * constHueColorSpaceMat[4]) +
		(chromaCr * constHueColorSpaceMat[5]);

	*blue = (luma     * constHueColorSpaceMat[6]) +
		(chromaCb * constHueColorSpaceMat[7]) +
		(chromaCr * constHueColorSpaceMat[8]);
}

__device__ static unsigned int RGBA_pack_10bit(float red, float green, float blue, unsigned int alpha)
{
	unsigned int ARGBpixel = 0;

	// Clamp final 10 bit results
	red = ::fmin(::fmax(red, 0.0f), 1023.f);
	green = ::fmin(::fmax(green, 0.0f), 1023.f);
	blue = ::fmin(::fmax(blue, 0.0f), 1023.f);

	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((unsigned int)blue >> 2) |
		(((unsigned int)green >> 2) << 8) |
		(((unsigned int)red >> 2) << 16) |
		(unsigned int)alpha);

	return ARGBpixel;
}

// CUDA kernel for outputting the final ARGB output from NV12

#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF

extern "C" __global__ void NV12ToARGB(const unsigned char* srcImage, size_t nSourcePitch,
	unsigned int* dstImage, size_t nDestPitch,
	unsigned int width, unsigned int height)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	const int y = blockIdx.y *  blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values

	unsigned int yuv101010Pel[2];

	yuv101010Pel[0] = (srcImage[y * nSourcePitch + x]) << 2;
	yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

	const size_t chromaOffset = nSourcePitch * height;

	const int y_chroma = y >> 1;

	if (y & 1)  // odd scanline ?
	{
		unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x];
		unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

		if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
		{
			chromaCb = (chromaCb + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x] + 1) >> 1;
			chromaCr = (chromaCr + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x + 1] + 1) >> 1;
		}

		yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}
	else
	{
		yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}

	// this steps performs the color conversion
	unsigned int yuvi[6];
	float red[2], green[2], blue[2];

	yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
	yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
	yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
	yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
	yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	// YUV to RGB Transformation conversion
	YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
	YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

	// Clamp the results to RGBA

	const size_t dstImagePitch = nDestPitch >> 2;

	dstImage[y * dstImagePitch + x] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
	dstImage[y * dstImagePitch + x + 1] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
}

extern "C" __global__ void Luminance(unsigned char *srcImage, size_t nSourcePitch, int left, int right, int top, int bottom, long long *result)
{
	long long total = 0;
	for (int x=left; x<=right; x++)
		for (int y=top; y<=bottom; y++)
			total += srcImage[y * nSourcePitch + x];
	*result = total;
}

