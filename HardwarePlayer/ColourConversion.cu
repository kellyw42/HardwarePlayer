
#include <hip/hip_runtime.h>
//nvcc --cubin -O3 -gencode=arch=compute_30,code=sm_30  ColourConversion.cu -odir ..\x64\Debug

extern "C" __global__ void NV12ToARGB(unsigned char *srcImage, size_t nSourcePitch, unsigned int *dstImage, size_t nDestPitch, int width, int height)
{
	int x, y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		//printf("%d,%d", x, y);
		return;
	}

	unsigned char lum = srcImage[y * nSourcePitch + x];

	dstImage[y * nDestPitch + x] = (0xFF << 24) | (lum << 16) | (lum << 8) | lum;
}

extern "C" __global__ void Luminance(unsigned char *srcImage, size_t nSourcePitch, int left, int right, int top, int bottom, long long *result)
{
	long long total = 0;
	for (int x=left; x<=right; x++)
		for (int y=top; y<=bottom; y++)
			total += srcImage[y * nSourcePitch + x];
	*result = total;
}

