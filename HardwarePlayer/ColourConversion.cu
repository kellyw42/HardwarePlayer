
#include <hip/hip_runtime.h>
//#include <stdio.h>
//#include <stdlib.h>
//#include <string.h>
//#include <cuda.h>

//nvcc --cubin -G -gencode=arch=compute_30,code=sm_30  ColourConversion.cu

extern "C" __global__ void NV12ToARGB(unsigned char *srcImage, size_t nSourcePitch, unsigned int *dstImage, size_t nDestPitch, int width, int height)
{
	int x, y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		//printf("%d,%d", x, y);
		return;
	}

	unsigned char lum = srcImage[y * nSourcePitch + x];

	dstImage[y * nDestPitch + x] = (0xFF << 24) | (lum << 16) | (lum << 8) | lum;
}

