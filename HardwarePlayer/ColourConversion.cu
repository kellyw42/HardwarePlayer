
#include <hip/hip_runtime.h>
// nvcc --cubin -O3 -gencode=arch=compute_75,code=sm_75  ColourConversion.cu
// Note: path to ColourConversion.cubin is hardcoded in VideoConvert.h

extern "C" __global__ void NV12ToGrayScale(unsigned char *srcImage, size_t nSourcePitch, unsigned int *dstImage, size_t nDestPitch, int width, int height)
{
	int x, y;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		//printf("%d,%d", x, y);
		return;
	}

	unsigned char lum = srcImage[y * nSourcePitch + x];

	dstImage[y * nDestPitch + x] = (0xFF << 24) | (lum << 16) | (lum << 8) | lum;
}

extern "C" __global__ void Copy(unsigned int* srcImage, unsigned int* dstImage, int centreX, int centreY, int dstX, int dstY)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= 100 || y >= 50) return;

	int dstPitch = 1920;
	int srcPitch = 1440;

	dstImage[(dstY * 64 + y) * dstPitch + (dstX * 128 + x)] = 
		srcImage[(centreY - 32 + y) * srcPitch + (centreX - 64 + x)];
}

__device__ static void YUV2RGB(const unsigned int* yuvi, float* red, float* green, float* blue)
{
	float constHueColorSpaceMat[9] = { 1.1644f, 0.0f, 1.596f, 1.1644f, -0.3918f, -0.813f, 1.1644f, 2.0172f, 0.0f };

	float luma, chromaCb, chromaCr;

	// Prepare for hue adjustment
	luma = (float)yuvi[0];
	chromaCb = (float)((int)yuvi[1] - 512.0f);
	chromaCr = (float)((int)yuvi[2] - 512.0f);

	// Convert YUV To RGB with hue adjustment
	*red = (luma     * constHueColorSpaceMat[0]) +
		(chromaCb * constHueColorSpaceMat[1]) +
		(chromaCr * constHueColorSpaceMat[2]);

	*green = (luma     * constHueColorSpaceMat[3]) +
		(chromaCb * constHueColorSpaceMat[4]) +
		(chromaCr * constHueColorSpaceMat[5]);

	*blue = (luma     * constHueColorSpaceMat[6]) +
		(chromaCb * constHueColorSpaceMat[7]) +
		(chromaCr * constHueColorSpaceMat[8]);
}

__device__ static unsigned int RGBA_pack_10bit(float red, float green, float blue, unsigned int alpha)
{
	unsigned int ARGBpixel = 0;

	// Clamp final 10 bit results
	red = ::fmin(::fmax(red, 0.0f), 1023.f);
	green = ::fmin(::fmax(green, 0.0f), 1023.f);
	blue = ::fmin(::fmax(blue, 0.0f), 1023.f);

	// Convert to 8 bit unsigned integers per color component
	ARGBpixel = (((unsigned int)blue >> 2) |
		(((unsigned int)green >> 2) << 8) |
		(((unsigned int)red >> 2) << 16) |
		(unsigned int)alpha);

	return ARGBpixel;
}

// CUDA kernel for outputting the final ARGB output from NV12

#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF

extern "C" __global__ void NV12ToARGB(const unsigned char* srcImage, size_t nSourcePitch,
	unsigned int* dstImageTop, unsigned int *dstImageBottom, size_t nDestPitch,
	unsigned int width, unsigned int height)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	const int y = blockIdx.y *  blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values

	unsigned int yuv101010Pel[2];
	yuv101010Pel[0] = (srcImage[y * nSourcePitch + x]) << 2;
	yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

	const size_t chromaOffset = nSourcePitch * height;

	int y_chroma = ((y >> 2) << 1) | (y & 1);

	unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x];
	unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

	yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
	yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
	yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

	unsigned int yuvi[6];
	float red[2], green[2], blue[2];

	yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
	yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
	yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
	yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
	yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
	yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

	YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
	YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

	unsigned int* dstImage = (y & 1) ? dstImageBottom : dstImageTop;
	int offset = (y >> 1) * width + x;
	dstImage[offset] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
	dstImage[offset + 1] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
}

extern "C" __global__ void FindFlash(unsigned int* image, int width, int height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	unsigned int* mem = &image[y * width + x];

	if (*mem == 0xFFFFFFFF) 
		*mem = 0xFF000000;

	//FF|red|green|blue
}

extern "C" __global__ void DrawComponents(unsigned int* image, int width, int height, unsigned int* componentLabels, unsigned int* sizes, int roiLeft, int roiTop, int roiWidth, int roiHeight, int step)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	if (x < roiLeft || x >= roiLeft + roiWidth || y < roiTop || y >= roiTop + roiHeight)
		return;

	int component = componentLabels[(y - roiTop) * step + (x - roiLeft) ];

	int left = componentLabels[(y - roiTop) * step + (x-1 - roiLeft)];
	int right = componentLabels[(y - roiTop) * step + (x+1 - roiLeft)];
	int top = componentLabels[(y-1 - roiTop) * step + (x - roiLeft)];
	int bottom = componentLabels[(y + 1 - roiTop) * step + (x - roiLeft)];

	int edge = component != left || component != right || component != top || component != bottom;
	
	if (component == 2048)
		image[y * width + x] = 0xFFFFFF00;
	else if (component == 4096)
		image[y * width + x] = 0xFFFF00FF;
	else if (component > 0 && sizes[component] > 1000 && edge)
        image[y * width + x] = 0xFF0000FF;

	//FF|red|green|blue
}

extern "C" __global__ void FinishLine3(const unsigned char* srcImage, size_t nSourcePitch,
	unsigned int* dstImageTop, unsigned int *dstImageBottom, size_t nDestPitch,
	unsigned int width, unsigned int height, float angle)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	const int y = blockIdx.y *  blockDim.y + threadIdx.y;

	if (y >= height)
		return;

	for (int dx=0; dx<width; dx+=2)
	{
	    int x = y * angle + dx;

		unsigned int yuv101010Pel[2];
		yuv101010Pel[0] = (srcImage[y * nSourcePitch + x]) << 2;
		yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

		const size_t chromaOffset = nSourcePitch * height;

		int y_chroma = ((y >> 2) << 1) | (y & 1);

		unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x];
		unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

		yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
		yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
		yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		unsigned int yuvi[6];
		float red[2], green[2], blue[2];

		yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
		yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
		yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);
		yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
		yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE)        & COLOR_COMPONENT_MASK);
		yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

		YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
		YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

		unsigned int* dstImage = (y & 1) ? dstImageBottom : dstImageTop;
		int offset = (y >> 1) * width + dx;
		dstImage[offset] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
		dstImage[offset + 1] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
	}
}

extern "C" __global__ void FinishLine2(unsigned int* dstImage, unsigned int width, unsigned int height, float top, float bottom, int y1, int y2, unsigned char* mask, int step)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	if (mask[y * step + x] > 0)
		return;

	double m = ((double)height) / (bottom - top);

	double dist = y - (m * (x - top));

	double LineWidth = 100;
	double LineBlur = 0;

	if (-LineWidth <= dist && dist <= LineWidth)
		dstImage[y * width + x] = 0xFFFF0000;
	else if (-LineWidth - LineBlur < dist && dist < -LineWidth)
	{
		double mix = (-LineWidth - dist) / LineBlur;
		int old = dstImage[y * width + x];
		int red = (old >> 16) & 0xFF;
		int green = (old >> 8) & 0xFF;
		int blue = old & 0xFF;
		red = red * mix + 0xFF * (1 - mix);
		dstImage[y * width + x] = (0xFF << 24) | (red << 16) | (green << 8) | blue;
	}
	else if (LineWidth < dist && dist < LineWidth + LineBlur)
	{
		double mix = (dist - LineWidth) / LineBlur;
		int old = dstImage[y * width + x];
		int red = (old >> 16) & 0xFF;
		int green = (old >> 8) & 0xFF;
		int blue = old & 0xFF;
		red = red * mix + 0xFF * (1 - mix);
		dstImage[y * width + x] = (0xFF << 24) | (red << 16) | (green << 8) | blue;
	}

	return;

	if (x >= width || x < bottom || y < y1 || y > y2)
		return;

	float dy = ((float)y) / height;
	int copyx = dy * bottom + (1 - dy) * top;
	int src = dstImage[y * width + copyx];
	int R = ((src >> 16) & 0xFF) - 180;
	int G = ((src >> 8) & 0xFF) - 73;
	int B = ((src >> 0) & 0xFF) - 100;
	float diff = (R * R + G * G + B * B);
	if (diff < 2000)
		dstImage[y * width + x] = 0xFF000000;
	else
		dstImage[y * width + x] = 0xFFFFFFFF;
}

extern "C" __global__ void FinishLine(unsigned int* dstImage, unsigned int width, unsigned int height, float top, float bottom, int y1, int y2, int*hits)
{
	int total = 0;
	for (int y = y1; y < y2; y++)
	{
		float dy = ((float)y) / height;
		int copyx = dy * bottom + (1 - dy) * top;
		int src = dstImage[y * width + copyx];
		int R = ((src >> 16) & 0xFF) - 180;
		int G = ((src >> 8) & 0xFF) - 73;
		int B = ((src >> 0) & 0xFF) - 100;
		float diff = (R * R + G * G + B * B);
		if (diff < 2000)
			total++;
	}
	*hits = total;
}


extern "C" __global__ void Luminance(unsigned char *srcImage, size_t nSourcePitch, int left, int right, int top, int bottom, long long *lum)
{
	long long topLum = 0, bottomLum = 0;
	for (int x = left; x <= right; x++)
		for (int y = top; y <= bottom; y++)
			if (y & 1)
				bottomLum += srcImage[y * nSourcePitch + x];
			else
				topLum += srcImage[y * nSourcePitch + x];
	lum[0] = topLum;
	lum[1] = bottomLum;
}

